#include <iostream>
#include <hip/hip_runtime.h>

// CUDA kernel for vector addition
__global__ void vector_add(const float* A, const float* B, float* C, int N) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < N)
        C[idx] = A[idx] + B[idx];
}

// Function for launching the kernel
void solve(const float* A, const float* B, float* C, int N) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    vector_add<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, N);
    hipDeviceSynchronize();  // Wait for kernel to finish
}

int main() {
    int N = 1 << 20; 

    // Allocate host memory
    float *h_A = new float[N];
    float *h_B = new float[N];
    float *h_C = new float[N];

    // Initialize host data
    for (int i = 0; i < N; ++i) {
        h_A[i] = i * 1.0f;
        h_B[i] = (N - i) * 1.0f;
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, N * sizeof(float));
    hipMalloc(&d_B, N * sizeof(float));
    hipMalloc(&d_C, N * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_A, h_A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel via `solve()`
    solve(d_A, d_B, d_C, N);

    // Copy result from device to host
    hipMemcpy(h_C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    // Verify result
    bool correct = true;
    for (int i = 0; i < N; ++i) {
        float expected = h_A[i] + h_B[i];
        if (fabs(h_C[i] - expected) > 1e-5) {
            std::cerr << "Mismatch at index " << i << ": got " << h_C[i] << ", expected " << expected << std::endl;
            correct = false;
            break;
        }
    }

    if (correct) std::cout << "Vector addition successful!" << std::endl;

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}
